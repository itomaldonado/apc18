
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>


__global__ void cuPi(float *sum, int nbin, float step)
{
  
    // Write your pi calculation kernel here
    return;
}


int main(void)
{
    int REAL_PI = 3.141592653589793238462643383;
    int NBINS = 10; // modify this to achieve better performance
    int STEP = 5;   // modify this to achieve better performance
    float* pi;

    hipMallocManaged(&pi, sizeof(float));

    //   Calculate Pi

    cuPi<<<1, 1>>>(pi, NBINS, STEP);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("Pi calculated as %f\n", *pi);

    float acc = REAL_PI - *pi;
    printf("Accuracy of Pi calculation %f\n", acc);
    
    // Free memory
    hipFree(pi);
    
    return 0;
}