
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

// kernels transpose a tile of TILE_DIM x TILE_DIM elements
// using a TILE_DIM x BLOCK_ROWS thread block, so that each thread
// transposes TILE_DIM/BLOCK_ROWS elements.
// TILE_DIM must be an integral multiple of BLOCK_ROWS
#define SIZE 10016
#define TILE_DIM 32
#define BLOCK_ROWS 8


__global__ void cuTranspose(int rows, int cols, const float *input, float *output) {

  __shared__ float tile[TILE_DIM][TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = x + (y)*rows;
  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = x + (y)*cols;
  
  for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
    tile[threadIdx.y+i][threadIdx.x] = input[index_in+i*rows];
  }
  __syncthreads();

  for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
    output[index_out + i*cols] = tile[threadIdx.x][threadIdx.y + i];
  }

  return;
}

int check(int nrows, int ncols, float* input, float* output) {
    for (int i = 0; i < nrows; i++){
        for(int j = 0; j < ncols; j++) {
            if (input[i*ncols + j] != output[j*ncols + i]) {
                printf("Input at (%i, %i) not equal to output at (%i, %i) with %f != %f\n", i, j, j, i, input[i*ncols + j], output[j*ncols + i]);
                return 0;
            }
        }
    }
    return 1;
}

int main(void)
{
  int r = SIZE;  // rows
  int c = SIZE;  // columns
  int N = r*c;    // total matrix entries.

  float *input, *output;

  dim3 dimGrid(r/TILE_DIM, c/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&input, N*sizeof(float));
  hipMallocManaged(&output, N*sizeof(float));

  // initialize input and output matrices on the host
  for (int i = 0; i < r; i++) {
      for(int j = i; j < c; j++) {
        input[i*c + j] = (float)r*i + r*1.0f;
        output[i*c + j] = 5.0f;
      }

  }

  // Transpose the matrix
  cuTranspose<<<dimGrid, dimBlock>>>(r, c, input, output);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  if (!check(r, c, input, output)) {
      printf("Matrix entries not equal\n"); 
  } else {
      printf("Matrix transpose successful\n");
  }

  // Free memory
  hipFree(input);
  hipFree(output);
  
  return 0;
}
