#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>


__global__ void cuTranspose(int rows, int cols, float *input, float *output)
{
  
    // Write your transpose kernel here
    return;
}

int check(int nrows, int ncols, float* input, float* output) {
    for (int i = 0; i < nrows; i++){
        for(int j = 0; j < ncols; j++) {
            if (input[i*ncols + j] != output[j*ncols + i]) {
                printf("Input at (%i, %i) not equal to output at (%i, %i) with %f != %f\n", i, j, j, i, input[i*ncols + j], output[j*ncols + i]);
                return 0;
            }
        }
    }
    return 1;
}

int main(void)
{
  int r = 10000; // rows
  int c = 10000;  // columns
  int N = r*c;   // total matrix entries.

  float *input, *output;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&input, N*sizeof(float));
  hipMallocManaged(&output, N*sizeof(float));

  // initialize input and output matrices on the host
  for (int i = 0; i < r; i++) {
      for(int j = i; j < c; j++) {
        input[i*c + j] = (float)nrows*i + r*1.0f;
        output[i*c + j] = 5.0f;
      }

  }

//   Transpose the matrix
  cuTranspose<<<1, 1>>>(r, c, input, output);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  if (!check(r, c, input, output)) {
      printf("Matrix entries not equal\n"); 
  } else {
      printf("Matrix transpose successful\n");
  }

  // Free memory
  hipFree(input);
  hipFree(output);
  
  return 0;
}
