
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

// kernels transpose a tile of TILE_DIM x TILE_DIM elements
// using a TILE_DIM x BLOCK_ROWS thread block, so that each thread
// transposes TILE_DIM/BLOCK_ROWS elements.
// TILE_DIM must be an integral multiple of BLOCK_ROWS
#define TILE_DIM 32
#define BLOCK_ROWS 8


__global__ void cuTranspose(int rows, int cols, const float *input, float *output) {

  //int x = blockIdx.x * TILE_DIM + threadIdx.x;
  //int y = blockIdx.y * TILE_DIM + threadIdx.y;
  //int width = gridDim.x * TILE_DIM;

  int tile_size = blockDim.x ;
  int column = tile_size * blockIdx.x + threadIdx.x;
  int row = tile_size * blockIdx.y + threadIdx.y;
  if(column < cols && row < rows){
    output[column*cols + row] = input[column + row*rows];
  }

  /*
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
    output[x*width + (y+j)] = input[(y+j)*width + x];
  }
  */

  return;
}

int check(int nrows, int ncols, float* input, float* output) {
    for (int i = 0; i < nrows; i++){
        for(int j = 0; j < ncols; j++) {
            if (input[i*ncols + j] != output[j*ncols + i]) {
                printf("Input at (%i, %i) not equal to output at (%i, %i) with %f != %f\n", i, j, j, i, input[i*ncols + j], output[j*ncols + i]);
                return 0;
            }
        }
    }
    return 1;
}

int main(void)
{
  int r = 10000; // rows
  int c = 10000;  // columns
  int N = r*c;   // total matrix entries.

  float *input, *output;

  dim3 dimGrid(r/TILE_DIM, c/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&input, N*sizeof(float));
  hipMallocManaged(&output, N*sizeof(float));

  // initialize input and output matrices on the host
  for (int i = 0; i < r; i++) {
      for(int j = i; j < c; j++) {
        input[i*c + j] = (float)r*i + r*1.0f;
        output[i*c + j] = 5.0f;
      }

  }

  // Transpose the matrix
  printf("Before\n");
  cuTranspose<<<dimGrid, dimBlock>>>(r, c, input, output);
  printf("After\n");

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  if (!check(r, c, input, output)) {
      printf("Matrix entries not equal\n"); 
  } else {
      printf("Matrix transpose successful\n");
  }

  // Free memory
  hipFree(input);
  hipFree(output);
  
  return 0;
}
